#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>

/***********************************************************************Seperate Channel Kernel**************************************************************/

//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{
  // TODO
  //
  // NOTE: Be careful not to try to access memory that is outside the bounds of
  // the image. You'll want code that performs the following check before accessing
  // GPU memory:
  // Basically make int2 creates a structure of 2 fields and will initiliaze both of them..

  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);  //  it constructs a vector with value x, y., 
  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;	 //  
  const int absolute_image_position_x = thread_2D_pos.x;
  const int absolute_image_position_y = thread_2D_pos.y;
  if ( absolute_image_position_x >= numCols ||
       absolute_image_position_y >= numRows )
  {
      return;
  }
  redChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].x;
  greenChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].y;
  blueChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].z;
}


/****************************************************************Convolution Kernel*******************************************************************************/

__global__ void gaussian_blur(const unsigned char* const inputChannel,
                    unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth,const int s, int oRows, int oCols)
{
  // TODO
  
  // NOTE: Be sure to compute any intermediate results in floating point
  // before storing the final result as unsigned char.

  // NOTE: Be careful not to try to access memory that is outside the bounds of
  // the image. You'll want code that performs the following check before accessing
  // GPU memory:
  //
 
    
int x=blockIdx.x * blockDim.x + threadIdx.x;
    int y=blockIdx.y * blockDim.y + threadIdx.y;
    const int thread_1D_pos = y * oCols + x;


   if ( x>=oCols ||y>=oRows )
   {
       return;
   }


float sum=0.0f;
int kidx=0;
   for(int r=0; r<filterWidth;++r){
        for(int c=0; c<filterWidth;++c){
        
            int idx=(y*s+r)*numCols+x*s+c;
            
        float filter_value=filter[kidx++];
        sum+=filter_value*static_cast<float>(inputChannel[idx]);
   
        } 
    }
    outputChannel[thread_1D_pos]=sum;
}




/**************************************************************Recombine Channels Kernel***************************************************************/

__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       unsigned char * outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  //make sure we don't try and access memory outside the image
  //by having any threads mapped there return early

  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;


  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  //Alpha should be 255 for no transparency
  //uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = red+green+blue;

}




/*******************************************************************Pooling Layer******************************************************************************/

__global__ void pooling_layer(unsigned char* image,unsigned char* output_image ,int oRow,int oCol,int fsize,int stride)
{
	  
	int output_rows=((oRow-fsize)/stride +1);
	int output_columns=((oCol-fsize)/stride +1);
	int i,j;
	float sum=0;
	float mask=0;
	int column = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	if(row>=oRow || column>=oCol)
		return;

        int global_index=(row*output_columns)+column;

	int row_input=(row*fsize);
	int col_input=(column*fsize);

	if(row_input>=oRow-1 || col_input>=oCol-1)
		return;

       	
	for(i=0;i<fsize;i++)
	{
		for(j=0;j<fsize;j++)
		{
			sum= image[(row_input+i) *oCol + col_input+j];			
			if(sum>mask)
			{
				mask=sum;
			}
		}
	}


	output_image[global_index]=mask;
	printf("The global index is %d and values is %f\n", global_index,mask);

}
		


/***********************************************************************************************************************************************************
Functions-
-----------
void allocateMemoryAndCopyToGPU- Allocates Memory for 3 different channels and Kernel Fiter.
void seperate_channel- Seperates an image into 3 channels.
void conv_firstlayer()- Calls Convolution Kernel.


***********************************************************************************************************************************************************/




/***********************************************Allocate Memory*********************************************************************************************/

	//According to me Onumpixels should be allocated.


unsigned char *d_red, *d_green, *d_blue;
//float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage)
{

  int i;

  //allocate memory for the three different channels
  //original
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));


  //TODO:
  //Allocate memory for the filter on the GPU
  //Use the pointer d_filter that we have already declared for you
  //You need to allocate memory for the filter with hipMalloc
  //be sure to use checkCudaErrors like the above examples to
  //be able to tell if anything goes wrong
  //IMPORTANT: Notice that we pass a pointer to a pointer to hipMalloc
  //checkCudaErrors(hipMalloc(&d_filter, sizeof( float) * filterWidth * filterWidth));
  //TODO:
  //Copy the filter on the host (h_filter) to the memory you just allocated
  //on the GPU.  hipMemcpy(dst, src, numBytes, hipMemcpyHostToDevice);
  //Remember to use checkCudaErrors!

  //checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice));

}



/********************************************************Seperate Channels**********************************************************************************/


void seperate_channel(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                      const size_t numRows, const size_t numCols,
                      const int oRow,const int oCol)
{
	 
	const dim3 blockSize(32,2);
	const dim3 gridSize(oCol/blockSize.x+1,oRow/blockSize.y+1);


  //TODO: Launch a kernel for separating the RGBA image into different color channels
	separateChannels<<<gridSize, blockSize>>>(d_inputImageRGBA,


                                              numRows,
                                              numCols,
                                              d_red,
                                              d_green,
                                              d_blue);

	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
	
}



/*******************************************************Convolution Kernel Call*********************************************************************************/

void your_gaussian_blur(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA, const size_t numRows, const size_t numCols,
			float * Red,
		        float* Green,
		        float* Blue,
                        unsigned char * d_redBlurred,
                        unsigned char * d_greenBlurred,
                        unsigned char * d_blueBlurred,
                        const int filterWidth, const int tilesize, const int s, const int oRow, const int oCol)
{
  
	const dim3 blockSize(32,2);
	const dim3 gridSize(oCol/blockSize.x+1,oRow/blockSize.y+1);
 
	gaussian_blur<<<gridSize, blockSize>>>(d_red,
                                         d_redBlurred,
                                         numRows,
                                         numCols,
                                         Red,
                                         filterWidth,s,oRow,oCol);
  hipDeviceSynchronize(); 
  checkCudaErrors(hipGetLastError());
  gaussian_blur<<<gridSize, blockSize>>>(d_green,
                                         d_greenBlurred,
                                         numRows,
                                         numCols,
                                         Green,
                                         filterWidth,s,oRow,oCol);
  hipDeviceSynchronize(); 
 checkCudaErrors(hipGetLastError());
  gaussian_blur<<<gridSize, blockSize>>>(d_blue,
                                         d_blueBlurred,
                                         numRows,
                                         numCols,
                                         Blue,
                                         filterWidth,s,oRow,oCol);

  
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());


}

//	I need to copy the output to an auxillary array
/*******************************************************Recombine Channels*********************************************************************************/

void recombine_channels(unsigned char *d_outputImageRGBA,
			unsigned char *d_redBlurred,
                        unsigned char *d_greenBlurred,
                        unsigned char *d_blueBlurred,
			const size_t numRows, const size_t numCols,
			const int oRow,const int oCol)
{

	 
	const dim3 blockSize(32,2);
	const dim3 gridSize(oCol/blockSize.x+1,oRow/blockSize.y+1);		


	recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,
                                             d_greenBlurred,
                                             d_blueBlurred,
                                             d_outputImageRGBA,
                                             numRows,
                                             numCols);
  	hipDeviceSynchronize();
        //checkCudaErrors(hipGetLastError());

	

}



	
/*************************************************************************************************************************************************************/

void pool(unsigned char *image, unsigned char *output, int oRow,int oCol,int fsize,int stride)
{
	


	int output_rows=((oRow-fsize)/stride) +1;
  	int output_columns=((oCol-fsize)/stride) +1;	
	int pool_pixels= (output_rows*output_columns);


	const dim3 blocksize(16,16);
	const dim3 gridsize(output_rows/blocksize.y +1,output_columns/blocksize.y +1);
	
	
	pooling_layer<<<gridsize,blocksize>>>(image,output,oRow,oCol,fsize,stride);



}














	




























 










